// Launch Configuration for Large Size 
// When my indexing exceeds 1024  in 1D 

#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void dkernel(unsigned *vector, unsigned vectorsize){
    unsigned id = blockIdx.x * blockDim.x + threadIdx.x; 
    if (id < vectorsize) 
    vector[id] = id; 
}

#define BLOCKSIZE 1024

int main(int nn, char *str[]){ // first arg nn denotes total number of args wjile running the program "./lec4 hello world" nn = 3
                              // second arg str is an array of args after program name {"lec4","hello", "world"}

    unsigned N = atoi(str[1]);
    unsigned *vector , *hvector; 
    hipMalloc(&vector, N * sizeof(unsigned));
    hvector = (unsigned*)malloc(N * sizeof(unsigned));

    unsigned nblocks = ceil((float)N/BLOCKSIZE); // N/BLOCKSIZE will give truncated value by default so no usage of ceil happens 
    printf("nblocks = %d\n", nblocks);

    dkernel<<<nblocks, BLOCKSIZE>>>(vector, N);
    hipMemcpy(hvector, vector, N * sizeof(unsigned), hipMemcpyDeviceToHost);
    for (unsigned i= 0; i < N; ++i){
        printf("%4d ", hvector[i]);
    }
    return 0; 
}