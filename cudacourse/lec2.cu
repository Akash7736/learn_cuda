// Parallelize this C code 

// #include <stdio.h>


// #define N 100

// int main(){

//     int i; 
//     for(int i = 0; i < N; ++i){
//         printf("%d\n", i*i);
//     }
//     return 0; 
// }

// ==============================================

// #include <stdio.h>


// #define N 100

// __global__ void kernel(){
//     int idx = threadIdx.x;
//     printf("%d\n", idx*idx);

// }

// int main(){

//     // int i; 
//     // for(i = 0; i < N; ++i){
//     //     printf("%d\n", i*i);
//     // }
//     kernel<<<1, N>>>();
//     cudaDeviceSynchronize();

//     return 0; 
// }



// *****************************************
// Parallelize the C program 

// #include <stdio.h>


// #define N 100

// int main(){

//     int a[N], i; 
//     for(int i = 0; i < N; ++i){
//         a[i] = i*i;
//     }
//     return 0; 
// }

// ==============================================

// #include <stdio.h>


// #define N 100

// __global__ void kernel(int *a){
//     int idx = threadIdx.x;
//     a[idx] =  idx*idx;

// }

// int main(){


//     int *a;
//     int *a_gpu; // a_gpu pointer variable declared on CPU
//     a = (int *) malloc(N*sizeof(int));
//     cudaMalloc((void **) &a_gpu, N * sizeof(int)); // Memory allocation is done on GPU and 
//     // its memory address is stored in a_gpu. now a_gpu is pointing to a memory in GPU

//     cudaMemcpy(a_gpu, a, N * sizeof(int), cudaMemcpyHostToDevice ); // arg order --> destination and source
//     // in cudaMemcpy we pass adresses, so we copy data from location with adress 'a' to location with adress 'a_gpu'
//     // cudaMemcpy only accepts pointers as arguments
//     // the pointer variables declared in main function are stored in CPU but those declared inside kernal function are stored in GPU
//     kernel<<<1, N>>>(a_gpu);
//     // cudaDeviceSynchronize();
//     cudaMemcpy(a, a_gpu, N * sizeof(int), cudaMemcpyDeviceToHost );
//     // We dont need cudaDeviceSynchronize() if we use cudaMemcpy() the execution is blocked and code waits to finish the kernal
//     // cudaMemcpy() transfer happemns through PCI express bus which has a limited bandwidth so it can be a bottleneck if not carefully used
//     for(int i =0; i < N; i++){
//         printf("%d\n", a[i]);
//     }

//     return 0; 
// }


// ***************************************************************************************************

// Write a CUDA program to initialize  an array of size 32 to all zeros in parallel.
// change the array size to 1024 
// Create another kernel that adds i to array[i]
// Change the array size to 8000
// Check if answer to problem 3 still works 


#include <hip/hip_runtime.h>
#include <stdio.h>
#define N 8000
__global__ void kernel(int *arr){
    int idx = blockIdx.x*1024 + threadIdx.x; 
    if (idx<8000) {
    arr[idx] =  idx;
    }

}

int main(){
    int *darr;
    // harr = (int *)malloc(N*sizeof(int));
    int harr[N];
    hipMalloc(&darr, N * sizeof(int));
    // cudaMemcpy(darr, harr, N * sizeof(int),cudaMemcpyHostToDevice); // Not necessary  here as array is initialized on GPU
    kernel<<<8,1024>>>(darr);
    hipMemcpy(harr, darr, N*sizeof(int), hipMemcpyDeviceToHost);
    for(int i=0; i<N; i++){
        printf("%d\n", harr[i]);
    }
    hipFree(darr);
    return 0;
}

// when array size is 8000 with kernel function kernel<<<1,N>>>(darr); thenit will give garbage values at end
// threadblock - there is limit for the second term in kernel function call which is 1024. 

// ********************************************************************

