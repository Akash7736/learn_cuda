#include <stdio.h>
#include <hip/hip_runtime.h>

int main() {
    hipDeviceProp_t prop;
    int device;

    // Get the current device ID
    hipGetDevice(&device);

    // Get the properties of the current device
    hipGetDeviceProperties(&prop, device);

    // Print device name and memory properties
    printf("Device Name: %s\n", prop.name);
    printf("Shared Memory Per Block: %lu bytes\n", prop.sharedMemPerBlock);
    printf("Shared Memory Per Multiprocessor: %lu bytes\n", prop.sharedMemPerMultiprocessor);
    printf("Registers Per Block: %d\n", prop.regsPerBlock);
    printf("Max Threads Per Block: %d\n", prop.maxThreadsPerBlock);
    printf("Max Registers Per Thread: %d\n", prop.regsPerBlock / prop.maxThreadsPerBlock);
    printf("Total Constant Memory: %lu bytes\n", prop.totalConstMem);
    printf("Warp Size: %d threads\n", prop.warpSize);
    printf("Maximum Memory Pitch: %lu bytes\n", prop.memPitch);
    printf("Global Memory: %lu MB\n", prop.totalGlobalMem / (1024 * 1024));
    printf("Max Threads Dim (per block): %d x %d x %d\n",
           prop.maxThreadsDim[0], prop.maxThreadsDim[1], prop.maxThreadsDim[2]);
    printf("Max Grid Size (blocks): %d x %d x %d\n",
           prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);
    printf("Max Blocks per Dimension (Grid): %d x %d x %d\n", 
           prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);
    return 0;
}
