// Accesing Dimensions 


// #include <stdio.h>
// #include <cuda.h>

// __global__ void dkernel(){
//     if ( threadIdx.x ==0 && blockIdx.x ==0 && // I want to print only once that's why this condition 
//         threadIdx.y ==0 && blockIdx.y == 0 &&
//         threadIdx .z == 0 && blockIdx.z == 0 ){
//             printf("%d %d %d %d %d %d\n", gridDim.x, gridDim.y, gridDim.z,
//                                         blockDim.x, blockDim.y, blockDim.z);
//         }
// }

// int main() {
//     dim3 grid(2, 3, 4);
//     dim3 block(5, 6, 7); 
//     dkernel<<<grid, block>>>(); // Here grid is just a name, it still indicates the number of thread blocks (2 * 3 * 4) with which kernel is launched
//     // the name block indicates the number and dimension of threads being launched by the kernel 
//     // If instead of (2,3,4) i launch directly as 24 blocks, then it will be a 1D grid. It depends on use case what dimension we need to launch, there is no perfomance penalty in dimensions. 
//     cudaDeviceSynchronize();
//     return 0; 
// }

// // if i remove "threadIdx.x ==0 && blockIdx.x ==0" condition, it will print 5 * 2 times 


//  *********************************************************************

// Write the kernel to initialize the matrix to unique ids 

#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void dkernel(unsigned *matrix){
    int idx = threadIdx.x * blockDim.y + threadIdx.y;
    matrix[idx] = idx;

}

#define N 5
#define M 6
int main(){
    dim3 block(N, M, 1);
    unsigned *matrix, *hmatrix;
    hipMalloc(&matrix, N * M * sizeof(unsigned));
    hmatrix  = (unsigned*)malloc(N * M * sizeof(unsigned));
    dkernel<<<1, block>>>(matrix);
    hipMemcpy(hmatrix, matrix, N * M * sizeof(unsigned), hipMemcpyDeviceToHost);
    for (unsigned i = 0; i < N; ++i){
        for(unsigned j = 0; j < M; ++j){
            printf("%2d ", hmatrix[i * M + j]);
        }
        printf("\n");
    }
    return 0; 
}